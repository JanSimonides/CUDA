#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>
#include <time.h>

#define ROZMER_A 4096
#define ROZMER_B 4096


int* alokuj(int r, int s) {
	int* matica;
	matica = (int*)malloc(sizeof(int) * r * s);
	return matica;
}

void generuj(int* matica, int r, int s) {
	int i, j;
	for (i = 0; i < r; i++) {
		for (j = 0; j < s; j++) {
			matica[i * s + j] = rand() % 10;
		}
	}
}

void uvolni(int* matica) {
	hipFree(matica);
}

void novyVypis(int* matica, int r, int s) {
	int i, j;
	for (i = 0; i < r; i++) {
		for (j = 0; j < s; j++) {
			printf("%d ", matica[i * s + j]);
		}
		putchar('\n');
	}
	putchar('\n');
}

void pocitaj(int* a, int riadkyA, int stlpceA, int* b, int riadkyB, int stlpceB) {
	int i, j, k;
	if (stlpceA != riadkyB) {
		printf("Nemozem nasobit!\n");
	}
	else {
		printf("CPU\n");
		int* vysledok = alokuj(riadkyA, stlpceB);
		int sum = 0;
		for (i = 0; i < riadkyA; i++) {
			for (j = 0; j < stlpceB; j++) {
				for (k = 0; k < stlpceA; k++) {
					sum += a[i * stlpceA + k] * b[k * stlpceB + j];
				}
				vysledok[i * stlpceB + j] = sum;
				sum = 0;
			}
		}
		//novyVypis(vysledok, riadkyA, stlpceB);
		//return vysledok;
		uvolni(vysledok);
	}
}
__global__
void gpu_pocitaj(int* a, int riadkyA, int stlpceA, int* b, int riadkyB, int stlpceB, int* c) {
	if (stlpceA != riadkyB) {
		printf("Nemozem nasobit!\n");
	}
	else {
		int sum = 0;

		int col = blockDim.x *blockIdx.x + threadIdx.x;
		int row = blockDim.y * blockIdx.y + threadIdx.y;

		int k;
		if (col < riadkyA && row < stlpceB) {
			for (k = 0; k < stlpceA; k++) {
				sum += a[col*stlpceA+k] * b[k * stlpceB + row];
			}
			c[col * stlpceB + row] = sum;
			sum = 0;
		}
	}
}

int main()
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	srand(time(NULL));
	int* a, * b, * c;
	int* gpu_a, * gpu_b, * gpu_c;
	float ms = 0.0;

	a = alokuj(ROZMER_A, ROZMER_B);
	b = alokuj(ROZMER_B, ROZMER_B);
	c = alokuj(ROZMER_A, ROZMER_B);

	hipMalloc(&gpu_a, sizeof(int) * ROZMER_A * ROZMER_B);
	hipMalloc(&gpu_b, sizeof(int) * ROZMER_B * ROZMER_B);
	hipMalloc(&gpu_c, sizeof(int) * ROZMER_A * ROZMER_B);

	generuj(a, ROZMER_A, ROZMER_B);
	generuj(b, ROZMER_B, ROZMER_B);

	//novyVypis(a, ROZMER_A, ROZMER_B);
	//novyVypis(b, ROZMER_B, ROZMER_B);

	hipMemcpy(gpu_a, a, ROZMER_A * ROZMER_B * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpu_b, b, ROZMER_B * ROZMER_B * sizeof(int), hipMemcpyHostToDevice);

	uvolni(a);
	uvolni(b);

	dim3 threadsPerBlock(8, 8);
	dim3 numBlocks(ROZMER_A / threadsPerBlock.x, ROZMER_B / threadsPerBlock.y);

	hipEventRecord(start);
	gpu_pocitaj << <numBlocks, threadsPerBlock >> > (gpu_a, ROZMER_A, ROZMER_B, gpu_b, ROZMER_B, ROZMER_B, gpu_c);
	hipEventRecord(stop);
	hipDeviceSynchronize();
	//pocitaj(a,ROZMER_A,ROZMER_B,b,ROZMER_B,ROZMER_B);

	hipMemcpy(c, gpu_c, ROZMER_A * ROZMER_B * sizeof(int), hipMemcpyDeviceToHost);
	//novyVypis(c, ROZMER_A, ROZMER_B);

	hipFree(gpu_a);
	hipFree(gpu_b);
	hipFree(gpu_c);

	hipEventElapsedTime(&ms, start, stop);

	uvolni(c);
	printf("GPU = %.6f s\n", ms/1e3);
	return 0;
}
